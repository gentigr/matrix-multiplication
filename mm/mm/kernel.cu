#include "hip/hip_runtime.h"
#define _CRT_SECURE_NO_DEPRECATE

#include <stdio.h>

#include "hip/hip_runtime.h"
#include ""

#include "mm.h"

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int cuda_main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel << <1, size >> >(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}

void cuda_clear_info(hipDeviceProp_t *cdp)
{
    free(cdp);
}

int cuda_get_info(int *pDeviceCount, hipDeviceProp_t **pDeviceProp)
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    hipDeviceProp_t *cdp = (hipDeviceProp_t *)malloc(deviceCount * sizeof(hipDeviceProp_t));
    if (cdp == NULL) {
        printf("There is not enough memory for getting device information.\n");
        return -1;
    }

    for (int i = 0; i < deviceCount; i++)
        hipGetDeviceProperties(&cdp[i], i);

    *pDeviceCount = deviceCount;
    *pDeviceProp = cdp;

    return 0;
}

int cuda_print_info(int deviceCount, hipDeviceProp_t *deviceProp)
{
    printf("Device count: %d\n\n", deviceCount);

    for (int i = 0; i < deviceCount; i++)
    {
        printf("Device name: %s\n", deviceProp[i].name);
        printf("Total global memory: %Iu\n", deviceProp[i].totalGlobalMem);
        printf("Shared memory per block: %Iu\n", deviceProp[i].sharedMemPerBlock);
        printf("Registers per block: %d\n", deviceProp[i].regsPerBlock);
        printf("Warp size: %d\n", deviceProp[i].warpSize);
        printf("Memory pitch: %Iu\n", deviceProp[i].memPitch);
        printf("Max threads per block: %d\n", deviceProp[i].maxThreadsPerBlock);

        printf("Max threads dimensions: x = %d, y = %d, z = %d\n",
            deviceProp[i].maxThreadsDim[0],
            deviceProp[i].maxThreadsDim[1],
            deviceProp[i].maxThreadsDim[2]);

        printf("Max grid size: x = %d, y = %d, z = %d\n",
            deviceProp[i].maxGridSize[0],
            deviceProp[i].maxGridSize[1],
            deviceProp[i].maxGridSize[2]);

        printf("Clock rate: %d\n", deviceProp[i].clockRate);
        printf("Total constant memory: %Iu\n", deviceProp[i].totalConstMem);
        printf("Compute capability: %d.%d\n", deviceProp[i].major, deviceProp[i].minor);
        printf("Texture alignment: %d\n", deviceProp[i].textureAlignment);
        printf("Device overlap: %d\n", deviceProp[i].deviceOverlap);
        printf("Multiprocessor count: %d\n", deviceProp[i].multiProcessorCount);

        printf("Kernel execution timeout enabled: %s\n",
            deviceProp[i].kernelExecTimeoutEnabled ? "true" : "false");
    }

    return 0;
}

int cuda_get_and_print_info()
{
    int device_count;
    hipDeviceProp_t *cdp;

    if (cuda_get_info(&device_count, &cdp) != 0)
        return -1;

    if (cuda_print_info(device_count, cdp) != 0)
        return -1;

    cuda_clear_info(cdp);

    return 0;
}

hipError_t mmWithCuda(int device, float *c, const float *a, const float *b, unsigned int size);

#define BLOCK_SIZE 16

__global__ void mmKernel(float * a, float * b, int n, float * c)
{
    int bx = blockIdx.x, by = blockIdx.y;
    int tx = threadIdx.x, ty = threadIdx.y;
    int aBegin = n * BLOCK_SIZE * by;
    int aEnd = aBegin + n - 1;
    int bBegin = BLOCK_SIZE * bx;
    int aStep = BLOCK_SIZE, bStep = BLOCK_SIZE * n;
    float sum = 0.0f;
    for (int ia = aBegin, ib = bBegin; ia <= aEnd; ia += aStep, ib += bStep)
    {
        __shared__ float as[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float bs[BLOCK_SIZE][BLOCK_SIZE];
        as[ty][tx] = a[ia + n * ty + tx];
        bs[ty][tx] = b[ib + n * ty + tx];
        __syncthreads();
        for (int k = 0; k < BLOCK_SIZE; k++)
            sum += as[ty][k] * bs[k][tx];
        __syncthreads();
    }
    c[n * BLOCK_SIZE * by + BLOCK_SIZE * bx + n * ty + tx] = sum;
}

int convert_two_to_one_dimension(int size, float **mt, float **pmt)
{
    float *res = (float*)malloc(size * size * sizeof(float));
    if (res == NULL) {
        printf("There is not enough memory!\n");
        return -1;
    }

    for (int i = 0; i < size; i++)
        for (int j = 0; j < size; j++)
            res[i*size + j] = mt[i][j];

    *pmt = res;

    return 0;
}

int convert_one_to_two_dimension(int size, float *in, float **out)
{
    for (int i = 0; i < size; i++)
        for (int j = 0; j < size; j++)
            out[i][j] = in[i*size + j];

    return 0;
}

int cuda_mm(float **pa, float **pb, float **pc, unsigned int size)
{
    float *a, *b, *c;

    //TODO: Memory leaks
    if ((convert_two_to_one_dimension(size, pa, &a) != 0) ||
        (convert_two_to_one_dimension(size, pb, &b) != 0) ||
        (convert_two_to_one_dimension(size, pc, &c) != 0)) {
        return -1;
    }

    // Add vectors in parallel.
    hipError_t cudaStatus = mmWithCuda(0, c, a, b, size);
    if (cudaStatus != hipSuccess) {
        printf("mmWithCuda failed!");
        return 1;
    }

    if (convert_one_to_two_dimension(size, c, pc) != 0) {
        printf("Smth go wrong during convertation!\n");
        return -1;
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        printf("hipDeviceReset failed!");
        return 1;
    }

    free(a);
    free(b);
    free(c);

    return 0;
}

hipError_t mmWithCuda(int device, float *c, const float *a, const float *b, unsigned int size)
{
    float * adev, *bdev, *cdev;
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocks(size / threads.x, size / threads.y);
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system
    cudaStatus = hipSetDevice(device);
    if (cudaStatus != hipSuccess) {
        printf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return cudaStatus;
    }
    
    // Allocate GPU buffers for three matrixes
    cudaStatus = hipMalloc((void**)&adev, size * size * sizeof (float));
    if (cudaStatus != hipSuccess) {
        printf("hipMalloc failed!");
        return cudaStatus;
    }

    cudaStatus = hipMalloc((void**)&bdev, size * size * sizeof (float));
    if (cudaStatus != hipSuccess) {
        printf("hipMalloc failed!");
        hipFree(adev);
        return cudaStatus;
    }

    cudaStatus = hipMalloc((void**)&cdev, size * size * sizeof (float));
    if (cudaStatus != hipSuccess) {
        printf("hipMalloc failed!");
        hipFree(adev);
        hipFree(bdev);
        return cudaStatus;
    }

    // Copy input matrixes from host memory to GPU buffers.
    cudaStatus = hipMemcpy(adev, a, size * size * sizeof (float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        printf("hipMemcpy failed!");
        hipFree(adev);
        hipFree(bdev);
        hipFree(cdev);
        return cudaStatus;
    }

    cudaStatus = hipMemcpy(bdev, b, size * size * sizeof (float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        printf("hipMemcpy failed!");
        hipFree(adev);
        hipFree(bdev);
        hipFree(cdev);
        return cudaStatus;
    }

    // Launch a kernel on the GPU
    mmKernel << <blocks, threads >> > (adev, bdev, size, cdev);
    hipDeviceSynchronize();

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        printf("mmKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(adev);
        hipFree(bdev);
        hipFree(cdev);
        return cudaStatus;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        printf("hipDeviceSynchronize returned error code %d after launching mmKernel!\n", cudaStatus);
        hipFree(adev);
        hipFree(bdev);
        hipFree(cdev);
        return cudaStatus;
    }

    // Copy output matrix from GPU buffer to host memory
    cudaStatus = hipMemcpy(c, cdev, size * size * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        printf("hipMemcpy failed!");
    }

    hipFree(adev);
    hipFree(bdev);
    hipFree(cdev);

    return cudaStatus;
}